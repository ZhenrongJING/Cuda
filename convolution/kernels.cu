#include "hip/hip_runtime.h"
__global__ void padding(int const nchl, int const nrow, int const ncol,int const npad,
        float* img, float* imgPad){

    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;
    int const rowP = nrow + 2*npad;
    int const colP = ncol + 2*npad;

    if ( (i<ncol+2*npad) && (i<ncol+2*npad) ){
    for (int n=0; n<nchl; n++){
        int idxP = idxD(nchl, rowP, colP, n, j, i);
        if ( (i>npad-1 && i<ncol+npad) && (j>npad-1 && j<nrow+npad) ) {
            int idxI = idxD(nchl, nrow, ncol, n, j-npad, i-npad);
            imgPad[idxP] = img[idxI];
        } else {
            imgPad[idxP] = 0.f;
        }
    }
    }
}

__global__ void convl(int const nFilter, int const nchl, int const rowP, int const colP, int const rowF, int const colF, float* imgPad, float* imgR, float* filter){

    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;

    if (i<colP-colF && j<rowP-colF){
        for (int c=0; c<nchl; c++){
            int idxR = idxD(nchl, rowP-rowF, colP-colF, c, i, j); 
            imgR[idxR] = 0.0f;
            for (int ii=0; ii<rowF; ii++){
                for (int jj=0; jj<colF; jj++){
                    int idxF = idxD4(nFilter, nchl, rowF, colF, 0, c, ii, jj);
                    int idxP = idxD(nchl, rowP, colP, c, i+ii, j+jj);
                    imgR[idxR] += imgPad[idxP]*filter[idxF]; 
                }
            } 
        }
    }

}
