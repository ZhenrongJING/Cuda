
#include <hip/hip_runtime.h>
#include<stdio.h>

struct dataElem {
    int val;
    char* name;
};

__global__ void kernal(dataElem* e){
    printf("From the device %s\n", e->name);
};

int main() {

    dataElem* e;
    hipMallocManaged((void**)&e, sizeof(dataElem) );
    e->val = 10;

    hipMallocManaged((void**)&(e->name), sizeof(char)*(strlen("hello")+1), 0);
    strcpy(e->name, "hello");

    kernal<<<1,1>>>(e);

    printf("From the host %s\n", e->name);
    return 0;
}
