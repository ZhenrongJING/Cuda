
#include <hip/hip_runtime.h>
//__constant__ float filter[CHN][COL_F][ROW_F];

__global__ void convl(int const colR, int const rowR, float const* filter, float const* img, float* imgR){

    int const i= blockDim.y*blockIdx.y + threadIdx.y;
    int const j= blockDim.x*blockIdx.x + threadIdx.x;

    if (i<rowR && j<colR){
        int np = i*colR+j;
        imgR[np] = img[np];
    }

}
