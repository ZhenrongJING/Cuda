
#include <hip/hip_runtime.h>
#include <iostream>
using std::cerr;
using std::endl;

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl;}

#include<stdio.h>
#include<stdlib.h>

void init_mtx(float* mtx, int n_unknows){

    for(int i=0; i<n_unknows; i++){
        for (int j=0; j<(n_unknows+1); j++){
            int mp = i*(n_unknows+1) + j;
            mtx[mp] = (float)(rand()%10);
        }
    }
}

void gauss_solver(float* mtx, int const n_unknows){
    for (int i=0; i<1; i++){
        for (int j=i+1; j<n_unknows; j++){
            int const mp = i*(n_unknows+1) + i;
            float ratio = mtx[j*(n_unknows+1)+i]/mtx[mp];
            for ( int k=0; k<n_unknows; k++){
                mtx[j*(n_unknows+1)+k] -= ratio*mtx[i*(n_unknows+1)+k];
            }
        } 
    }
}

void print_mtx(float* mtx, int n_r, int n_c){

    if (n_c>12 || n_c> 12) {
        printf("too large to be printed");
        return;
    }

    for(int i=0; i<n_r; i++){
        for(int j=0; j<n_c; j++){
            int mp = i*n_c + j;
            printf("%6.2f ", mtx[mp]);
        }
        printf("\n");
    }
    printf("-------------------------------------\n");
}

int main() {

    float* arg_mtx;
    float* h_mtx;
    int const n_unknows = 8;
    size_t mSize = n_unknows*(n_unknows+1)*sizeof(float);

    CUDA_CHECK(hipMallocManaged((void**)&arg_mtx, mSize));
    h_mtx = (float*)malloc(mSize); 

    init_mtx(h_mtx, n_unknows);
    print_mtx(h_mtx, n_unknows, n_unknows+1);
    gauss_solver(h_mtx, n_unknows);
    print_mtx(h_mtx, n_unknows, n_unknows+1);

    return 0;
}
