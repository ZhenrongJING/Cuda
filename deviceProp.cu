#include<stdio.h>
#include<iostream>
using namespace std;


int main(int argc, char* argv[]){

    hipDeviceProp_t property;

    hipGetDeviceProperties(&property, 0);

    cout << property.name << endl;
    cout << property.major << endl;
    cout << property.minor << endl;
    cout << property.totalGlobalMem << endl;
    cout << property.clockRate << endl;
    cout << property.sharedMemPerBlock << endl;
    cout << property.regsPerBlock << endl;
    cout << "warpSize" << endl;
    cout << property.warpSize << endl;
    cout << "Maximum thread" << endl;
    cout << property.maxThreadsPerMultiProcessor << endl;
    cout << "number of MP" << endl;
    cout << property.multiProcessorCount<< endl;

    return 0;
}
