
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel(){

    printf("hello world");
};

int main(){

    hipStream_t stream;
    hipStreamCreate(&stream);

    for(int i=0; i<5; i++){
        kernel<<<1, 1>>>();
    }

    hipStreamDestroy(stream);

    hipDeviceSynchronize();

    return 0;
}
