#include "hip/hip_runtime.h"
//__constant__ float filter[CHN][COL_F][ROW_F];

__global__ void convl(int const rowR, int const colR, float const* filter, float const* img, float* imgR){

    int const i= blockDim.y*blockIdx.y + threadIdx.y;
    int const j= blockDim.x*blockIdx.x + threadIdx.x;

    if (i<rowR && j<colR){
        int np = i*colR+j;
        imgR[np] = 0.0f;
        for(int ii=0; ii<ROW_F; ii++){
            int ix = i-ROW_F/2+ii;
            for(int jj=0; jj<COL_F; jj++){
                int iy = j-COL_F/2+jj;
                float tmp;
                if (ix<0 || ix>=rowR || iy< 0 || iy>=colR){
                    tmp = 0.0f;
                }else{
                    int id = ix*colR + iy;
                    tmp = img[id];
                }
                imgR[np] += filter[ii*COL_F+jj]*tmp;
            }
        }
    }

}
