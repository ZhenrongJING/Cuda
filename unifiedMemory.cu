
#include <hip/hip_runtime.h>
#include<stdio.h>

struct dataElem {
    int val;
    char* name;
};

__global__ void kernal(dataElem* e){
    printf("From the device %s\n", e->name);
};

int main() {

    dataElem* e;
    hipMallocManaged((void**)&e, sizeof(dataElem) );
    e->val = 10;

    hipMallocManaged((void**)&(e->name), sizeof(char)*(strlen("hello")+1) );
    strcpy(e->name, "hello");

    printf("From the host %s\n", e->name);

//    kernal<<<1,1>>>(e);
//    cudaDeviceSynchronize();
    return 0;
}
