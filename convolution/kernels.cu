#include "hip/hip_runtime.h"
//__constant__ float filter[CHN][COL_F][ROW_F];

__global__ void convl(int const colR, int const rowR, float const* filter, float const* img, float* imgR){

    int const i= blockDim.y*blockIdx.y + threadIdx.y;
    int const j= blockDim.x*blockIdx.x + threadIdx.x;

    if (i<rowR && j<colR){
        int np = i*colR+j;
        imgR[np] = 0.0f;
        for(int ii=0; ii<ROW_F; ii++){
            for(int jj=0; jj<COL_F; jj++){

                int id = i*colR + j; 
                imgR[np] = img[id];

            }
        }
    }

}
