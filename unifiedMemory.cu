
#include <hip/hip_runtime.h>
#include <iostream>
using std::cerr;
using std::endl;

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl;}

#include<stdio.h>

struct dataElem {
    int val;
    char* name;
};

__global__ void kernal(dataElem* e){
    printf("From the device %s\n", e->name);
};

int main() {

    dataElem* e;
    CUDA_CHECK(hipMallocManaged((void**)&e, sizeof(dataElem)));
    e->val = 10;

    hipMallocManaged((void**)&(e->name), sizeof(char)*(strlen("hello")+1) );
    strcpy(e->name, "hello");

    printf("From the host %s\n", e->name);

//    kernal<<<1,1>>>(e);
//    cudaDeviceSynchronize();

    return 0;
}
