#include "hip/hip_runtime.h"
#include<cstdlib>
#include<stdio.h>
#include"help.h"

void initialize(float* mtx, int const nx, int const ny){
    int tmp = nx*ny;
    for(int i=0; i<tmp; i++){
        mtx[i] = rand()/(float)RAND_MAX;
    }
};

__global__ void sumMatrix2D2D(float* d_a, float* d_b, float* d_c, int const nx, int const ny){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int mp = j*ny+i;
    
    d_c[mp] = d_a[mp] + d_b[mp];
};

void computeGPU(float* h_a, float* h_b, float* h_c, int const mSize, int const nx, int const ny){

    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, mSize);
    hipMalloc((void**)&d_b, mSize);
    hipMalloc((void**)&d_c, mSize);
    hipMemcpy(d_a, h_a, mSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, mSize, hipMemcpyHostToDevice);

    int xBlock = 32;
    int yBlock = 32;

    dim3 block(xBlock, yBlock);
    dim3 grid(nx/xBlock, ny/yBlock);

    printf("run with block %d, %d", xBlock, yBlock);

    sumMatrix2D2D<<<grid, block>>>(d_a, d_b, d_c, nx, ny);
    hipMemcpy(h_c, d_c, mSize, hipMemcpyDeviceToHost);

    for (int i=0; i<nx*ny; i++){
        if ( abs(h_c[i] - (h_a[i] + h_b[i])) > 1e-4 ) {
            printf("2D2D");
            printf("%8.5f, %8.5f, %8.5f, %d \n", h_a[i], h_b[i], h_c[i], i);
            break;
        }
    }

    return;
}
