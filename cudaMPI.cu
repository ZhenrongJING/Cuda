
#include <hip/hip_runtime.h>


void initialisation_cuda()
{
    char* local_rank_env;
    int local_rank;
    hipError_t cudaRet;
 
     /* Recovery of the local rank of the process via the environment variable
        set by Slurm, as  MPI_Comm_rank cannot be used here because this routine
        is used BEFORE the initialisation of MPI*/
    local_rank_env = getenv("SLURM_LOCALID");
 
    if (local_rank_env) {
        local_rank = atoi(local_rank_env);
        /* Define the GPU to use for each MPI process */
        cudaRet = hipSetDevice(local_rank);
        if(cudaRet != hipSuccess) {
            printf("Erreur: hipSetDevice has failed\n");
            exit(1);
        }
    } else {
        printf("Error : impossible to determine the local rank of the process\n");
        exit(1);
    }
}

