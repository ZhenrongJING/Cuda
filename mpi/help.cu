#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<stdio.h>
#include"help.h"
using std::cerr;
using std::endl;

#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl;}

void initialize(float* mtx, int const nx, int const ny){
    int tmp = nx*ny;
    for(int i=0; i<tmp; i++){
        mtx[i] = rand()/(float)RAND_MAX;
    }
};

__global__ void sumMatrix2D2D(float* d_a, float* d_b, float* d_c, int const nx, int const ny){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int mp = j*ny+i;
    
    d_c[mp] = d_a[mp] + d_b[mp];
};

void computeGPU(float* h_a, float* h_b, float* h_c, int const mSize, int const nx, int const ny){

    float* d_a;
    float* d_b;
    float* d_c;
    CUDA_CHECK (hipMalloc((void**)&d_a, mSize));
    CUDA_CHECK (hipMalloc((void**)&d_b, mSize));
    CUDA_CHECK (hipMalloc((void**)&d_c, mSize));
    CUDA_CHECK (hipMemcpy(d_a, h_a, mSize, hipMemcpyHostToDevice));
    CUDA_CHECK (hipMemcpy(d_b, h_b, mSize, hipMemcpyHostToDevice));

    int xBlock = 32;
    int yBlock = 32;

    dim3 block(xBlock, yBlock);
    dim3 grid(nx/xBlock, ny/yBlock);

    printf("run with block %d, %d", xBlock, yBlock);

    sumMatrix2D2D<<<grid, block>>>(d_a, d_b, d_c, nx, ny);
    CUDA_CHECK (hipMemcpy(h_c, d_c, mSize, hipMemcpyDeviceToHost));

    for (int i=0; i<nx*ny; i++){
        if ( abs(h_c[i] - (h_a[i] + h_b[i])) > 1e-4 ) {
            printf("2D2D");
            printf("%8.5f, %8.5f, %8.5f, %d \n", h_a[i], h_b[i], h_c[i], i);
            break;
        }
    }

    return;
}
