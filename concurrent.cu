#define NSTREAM 4

#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void addVec(int* a, int* b, int* c, int const len){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i<len) c[i] = a[i] + b[i];
};

int main(){

    int const totalLen = 1<<16;
    int const mSize = totalLen*sizeof(int);

    int* h_a;
    int* h_b;
    int* h_c;

    hipHostAlloc((void**)&h_a, mSize, hipHostMallocDefault);
    hipHostAlloc((void**)&h_b, mSize, hipHostMallocDefault);
    hipHostAlloc((void**)&h_c, mSize, hipHostMallocDefault);

    for (int i=0; i<totalLen; i++){
        h_a[i] = i;
        h_b[i] = totalLen - i;
    }

    int* d_a;
    int* d_b;
    int* d_c;

    hipMalloc((void**)&d_a, mSize);
    hipMalloc((void**)&d_b, mSize);
    hipMalloc((void**)&d_c, mSize);

    int const lenPerStream = totalLen/NSTREAM;
    int const mSizePerStream = mSize/NSTREAM;

    hipStream_t lsStream[NSTREAM];

    for (int i=0; i<NSTREAM; i++){
        hipStreamCreate(&lsStream[i]);
    }

    int const block = 256;
    int const grid = lenPerStream/block;

    for (int i=0; i<NSTREAM; i++){
        int offset = i*lenPerStream;
        hipMemcpyAsync(&d_a[offset], &h_a[offset], mSizePerStream, hipMemcpyHostToDevice, lsStream[i]);
        hipMemcpyAsync(&d_b[offset], &h_b[offset], mSizePerStream, hipMemcpyHostToDevice, lsStream[i]);
        addVec<<<grid, block, 0, lsStream[i]>>>(&d_a[offset], &d_b[offset], &d_c[offset], lenPerStream);
        hipMemcpyAsync(&h_c[offset], &d_c[offset], mSizePerStream, hipMemcpyDeviceToHost, lsStream[i]);
    }

    for (int i=0; i<NSTREAM; i++){
        hipStreamSynchronize(lsStream[i]);
    }

    for (int i=0; i<totalLen; i++){
        if (h_c[i]!=totalLen) {
            printf("error, %d, %d \n", h_c[i], i);
            break;
        }
    }

    for (int i=0; i<NSTREAM; i++){
        hipStreamDestroy(lsStream[i]);
    }

    return 0;
}
