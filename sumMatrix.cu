
#include <hip/hip_runtime.h>
#include<cstdlib>
#include<stdio.h>

void initialize(float* mtx, int const nx, int const ny){
    int tmp = nx*ny;
    for(int i=0; i<tmp; i++){
        mtx[i] = rand()/(float)RAND_MAX;
    }

};

__global__ void sumMatrix2D2D(float* d_a, float* d_b, float* d_c, int const nx, int const ny){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int mp = i*ny+j;
    
    d_c[mp] = d_a[mp] + d_b[mp];
};

__global__ void sumMatrix1D1D(float* d_a, float* d_b, float* d_c, int const nx, int const ny){

    int i = blockIdx.x;
    for (; i < nx; i += gridDim.x){
        int j = threadIdx.x; 
        for (; j < ny; j += blockDim.x){
            int mp = i*ny + j;
            d_c[mp] = d_a[mp] + d_b[mp];
        }
    }
};

int main(int argc, char **argv){
    int const nx = 1<<14;
    int const ny = 1<<14;
    size_t mSize = nx*ny*sizeof(float);

    float* h_a;
    h_a = (float*)malloc(mSize);
    float* h_b;
    h_b = (float*)malloc(mSize);
    float* h_c;
    h_c = (float*)malloc(mSize);
    initialize(h_a, nx, ny);
    initialize(h_b, nx, ny);


    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, mSize);
    hipMalloc((void**)&d_b, mSize);
    hipMalloc((void**)&d_c, mSize);
    hipMemcpy(d_a, h_a, mSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, mSize, hipMemcpyHostToDevice);

    int xBlock = 32;
    int yBlock = 16;
    if(argc > 1) xBlock = atoi(argv[0]);
    if(argc > 2) xBlock = atoi(argv[1]);
    dim3 block(xBlock, yBlock);
    dim3 grid(nx/xBlock, ny/yBlock);

    printf("run with block %d, %d", xBlock, yBlock);

    sumMatrix2D2D<<<grid, block>>>(d_a, d_b, d_c, nx, ny);
    hipMemcpy(h_c, d_c, mSize, hipMemcpyDeviceToHost);

    for (int i=0; i<nx*ny; i++){
        if ( abs(h_c[i] - (h_a[i] + h_b[i])) > 1e-4 ) {
            printf("2D2D");
            printf("%8.5f, %8.5f, %8.5f, %d \n", h_a[i], h_b[i], h_c[i], i);
            break;
        }
    }

    sumMatrix1D1D<<<128, 128>>>(d_a, d_b, d_c, nx, ny);
    hipMemcpy(h_c, d_c, mSize, hipMemcpyDeviceToHost);

    for (int i=0; i<nx*ny; i++){
        if ( abs(h_c[i] - (h_a[i] + h_b[i])) > 1e-4 ) {
            printf("1D1D");
            printf("%8.5f, %8.5f, %8.5f, %d \n", h_a[i], h_b[i], h_c[i], i);
            break;
        }
    }

    return 0;
}
