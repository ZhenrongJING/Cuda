#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <opencv2/opencv.hpp>

#include "index.hpp"
#include "kernels.cu"
using namespace cv;
using namespace std;


int main(int argc, char** argv )
{
    if ( argc != 2 )
    {
        printf("usage: DisplayImage.out <Image_Path>\n");
        return -1;
    }

    Mat image;
    image = imread( argv[1], 1 );
    Mat imageFloat;
    image.convertTo(imageFloat, CV_32FC3, 1.0/255.0);

    if ( !image.data )
    {
        printf("No image data \n");
        return -1;
    }

    printf( " image type %d\n", image.type() );
    printf( " image type %d\n", imageFloat.type() );

    int const nrow = imageFloat.rows;
    int const ncol = imageFloat.cols;
    int const nchl = imageFloat.channels();
    printf( " size of the image %d times  %d times  %d\n", nchl, nrow, ncol);

    float *h_img;
    h_img = new float[nchl*nrow*ncol];

    int np = 0;
    for (int c=0; c<nchl; c++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                np = c*(nrow*ncol)+i*ncol+j;
                h_img[np] = imageFloat.at<Vec3f>(i,j)[c]; 
            }
        }
    }

    float* d_img;
    int nElem;
    nElem = nchl*nrow*ncol;
    hipMalloc((void**)&d_img, nElem*sizeof(float));
    hipMemcpy(d_img, h_img, nElem*sizeof(float), hipMemcpyHostToDevice);

    int const npad = 16;
    int const rowP= nrow+2*npad; 
    int const colP= ncol+2*npad; 

    float* h_imgPad;
    nElem = nchl*rowP*colP;
    h_imgPad = new float[nElem];
    for (int i=0; i<nElem; i++)
        h_imgPad[i] = 0.f;

    for (int c=0; c<nchl; c++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                h_imgPad[idx(nchl, rowP, colP, c, i+npad, j+npad)]= h_img[c*(nrow*ncol)+i*ncol+j]; 
            }
        }
    }

    float* d_imgPad;
    nElem = nchl*rowP*colP;
    hipMalloc((void**)&d_imgPad, nElem*sizeof(float));
    int blockX=32;
    int blockY=32;
    dim3 block(blockX, blockY);
    int gridX = colP/32 + 1;
    int gridY = rowP/32 + 1;
    dim3 grid(gridX, gridY);
    padding<<<block,grid>>>(nchl, nrow, ncol, npad, d_img, d_imgPad);

    float* test;
    nElem = nchl*rowP*colP;
    test = new float[nElem];
    hipMemcpy(test, d_imgPad, nElem*sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<nElem; i++)
        if ( abs( test[i] - h_imgPad[i]) > 0.0001 ) {
            cout << i << " wrong" << endl;
            break;
        };

/*

    int const colF=2*npad, rowF=2*npad, nFilter=2;
    nElem = nFilter*nchl*rowF*colF;
    float *h_filter;
    h_filter = new float[nElem];
    for (int i=0; i<nElem; i++)
        h_filter[i] = 0.f;

    for (int n=0; n<nFilter; n++){
        for(int c=0; c<nchl; c++){
            for (int i=0; i<rowF; i++){
                for (int j=0; j<colF; j++){
                    h_filter[idx(nFilter,nchl,rowF,colF,n,c,i,j)] = rand()/(RAND_MAX+0.f); 
                }
            } 
        }
    }

    int const rowR= nrow+2*npad-rowF; 
    int const colR= ncol+2*npad-colF; 
    nElem = nFilter*nchl*rowR*colR;
    float* imageR;
    imageR = new float[nElem];



    float* d_filter;
    nElem = nFilter*nchl*rowF*colF;
    hipMalloc((void**)&d_filter, nElem*sizeof(float));
    hipMemcpy(d_filter, h_filter, nElem*sizeof(float), hipMemcpyHostToDevice);

    float* d_imgR;
    nElem = nFilter*nchl*rowR*colR;
    hipMalloc((void**)&d_imgR, nElem*sizeof(float));

    gridX = colR/32 + 1;
    gridY = rowR/32 + 1;
    dim3 grid1(gridX,gridY);

    convl<<<block, grid1>>>(nFilter, nchl, rowP, colP, rowF, colF, d_imgPad, d_imgR, d_filter);

    for (int n=0; n<nFilter; n++){
        for(int c=0; c<nchl; c++){
            for (int i=0; i<rowR; i++){
                for (int j=0; j<colR; j++){

                    imageR[idx(nFilter,nchl,rowR,colR,n,c,i,j)] = 0.0;
                    for(int ii=0; ii<rowF; ii++){
                        for(int jj=0; jj<colF; jj++){
                            imageR[idx(nFilter,nchl,rowR,colR,n,c,i,j)] +=
                                h_filter[idx(nFilter,nchl,rowF,colF,n,c,ii,jj)]
                                *h_imgPad[idx(nchl, rowP, colP, c, i+ii, j+jj)];
                        }
                    }

                }
            }
        }
    }

    for (int c=0; c<nchl; c++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                imageFloat.at<Vec3f>(i,j)[c] = imageR[idx(nFilter,nchl,rowR,colR,0,c,i,j)]/250.;
            }
        }
    }
    namedWindow("Display Image", WINDOW_AUTOSIZE );
    imshow("Display Image", imageFloat);
    waitKey(0);

*/
    return 0;
}

