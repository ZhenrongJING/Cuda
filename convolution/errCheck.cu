#include "errCheck.hpp"
#include <stdio.h>

void checkKernelLaunch(hipError_t* err) {
    if ( *err != hipSuccess ) {
        printf("CUDA Error: %s\n", hipGetErrorString(*err));
        exit(-1);
    }
}
