#include "hip/hip_runtime.h"
//__constant__ float filter[CHN][COL_F][ROW_F];

__global__ void convl(int const colR, int const rowR, float const* filter, float const* img, float* imgR){

    int const i= blockDim.y*blockIdx.y + threadIdx.y;
    int const j= blockDim.x*blockIdx.x + threadIdx.x;

    if (i<rowR && j<colR){
        int np = i*colR+j;
        imgR[np] = 0.0f;
        for(int ii=0; ii<ROW_F; ii++){
            for(int jj=0; jj<COL_F; jj++){
                int id = (i-ROW_F/2 +ii)*colR + (j-COL_F/2+jj); 
                float tmp;
                if (id<0 ||id>colR*rowR-1){
                    tmp = 0.0f;
                }else{
                    tmp = img[id];
                }
                imgR[np] += filter[ii*COL_F+jj]*tmp;
            }
        }
    }

}
