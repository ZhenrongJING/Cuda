
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernal1(float* d_a){
    float a;
    float b;
    a = b = 0.0;

    int nid = blockDim.x*blockIdx.x + threadIdx.x;
    if (nid%2 == 0){
        a = 100.0;
    }else{
        b = 200.0;
    }
    d_a[nid] = a+b;
};

int main(int argc, char** argv){
    int const n = 1<<16;
    int mSize = n*sizeof(float);

    float* d_a;
    hipMalloc((void**)&d_a, mSize);

    int xBlock = 256;
    kernal1<<<n/xBlock, xBlock>>>(d_a);

    hipDeviceSynchronize();
    return 0;
}
