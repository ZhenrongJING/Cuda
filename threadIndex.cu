
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void showCpy(float* d_a, int const nx, int const ny){
    unsigned int i = threadIdx.x;
    if(i<nx*ny) 
        printf("%d, %5.2f \n", i, d_a[i]); 
}

int main(){
    int const nx = 1<<4;
    int const ny = 1<<4;

    float h_a[nx][ny];
    for(int i=0; i<nx; i++){
        for(int j=0; j<ny; j++){
            h_a[i][j] = (float)i + ((float)j)/100;
        }
    }

    float* d_a;
    size_t mSize = nx*ny*sizeof(float);
    hipMalloc((void**)&d_a, mSize); 

    hipMemcpy(d_a, h_a, mSize, hipMemcpyHostToDevice);

    dim3 grid(1, 1);
    dim3 block(nx*ny, 1);

    showCpy<<<grid, block>>>(d_a, nx, ny);

/*    for(int i=0; i<nx; i++){
        for(int j=0; j<ny; j++){
           printf("%5.2f ", h_a[i][j]); 
        }
        printf("\n");
    }
*/

    hipDeviceSynchronize();
    return 0;
}
