#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <opencv2/opencv.hpp>

#define ROW_F 7
#define COL_F 7
#define CHN   3

#include "index.hpp"
#include "kernels.cu"
using namespace cv;
using namespace std;


int main(int argc, char** argv )
{
    if ( argc != 2 )
    {
        printf("usage: DisplayImage.out <Image_Path>\n");
        return -1;
    }

    Mat image;
    image = imread( argv[1], 1 );
    Mat imageFloat;
    image.convertTo(imageFloat, CV_32FC3, 1.0/255.0);

    if ( !image.data )
    {
        printf("No image data \n");
        return -1;
    }

    printf( " image type %d\n", image.type() );
    printf( " image type %d\n", imageFloat.type() );

    int const nrow = imageFloat.rows;
    int const ncol = imageFloat.cols;
    printf( " size of the image %d times  %d times  %d\n", CHN, nrow, ncol);

    float *h_img;
    h_img = new float[CHN*nrow*ncol];

    int np = 0;
    for (int c=0; c<CHN; c++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                np = c*(nrow*ncol)+i*ncol+j;
                h_img[np] = imageFloat.at<Vec3f>(i,j)[c]; 
            }
        }
    }

    int nElem;
    int const nFilter=2;
    nElem = CHN*nFilter*ROW_F*COL_F;
    float *h_filter;
    h_filter = new float[nElem];

    for(int c=0; c<CHN; c++){
        for (int n=0; n<nFilter; n++){
            for (int i=0; i<ROW_F; i++){
                for (int j=0; j<COL_F; j++){
                    h_filter[idx(CHN,nFilter,ROW_F,COL_F,c,n,i,j)] = rand()/(RAND_MAX+0.f); 
                }
            } 
        }
    }

    float* d_img;
    nElem = CHN*nrow*ncol;
    hipMalloc((void**)&d_img, nElem*sizeof(float));

    float* d_imgR;
    nElem = CHN*nFilter*nrow*ncol;
    hipMalloc((void**)&d_imgR, nElem*sizeof(float));

    float* d_filter[CHN];
    nElem = ROW_F*COL_F;
    
    for (int i=0;i<CHN;i++){
        hipMalloc((void**)&d_filter[i], nElem*sizeof(float));
    }

    for (int c=0;c<2;c++){
        int size = nrow*ncol;
        int offset = c*size;
        hipMemcpy(&d_img[offset], &h_img[offset], size*sizeof(float), hipMemcpyHostToDevice);

        for (int f=0;f<nFilter;f++){
            size = ROW_F*COL_F;
            offset = (c*nFilter+f)*size;
            hipMemcpy(d_filter[c], &h_filter[offset], size*sizeof(float), hipMemcpyHostToDevice);

            int const bx=32, by=32;
            int const gx=ncol/bx+1, gy=nrow/by+1;
            dim3 block(bx,by);
            dim3 grid (gx,gy);
            offset = (c*nFilter+f)*ncol*nrow;
            convl<<<block, grid>>>(nrow, ncol, d_filter[c], &d_img[c*nrow*ncol], &d_imgR[offset]);
        }
    }

/*
    float* test;
    nElem = CHN*nrow*ncol;
    test = new float[nElem];
    hipMemcpy(test, d_img, nElem*sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<nElem; i++){
        if ( abs(test[i] - h_img[i]) > 0.0001f ) {
            cout << i << ' ' << test[i] << ' ' << h_img[i] << endl;
            exit(0);
        }
    }

*/

    nElem = CHN*nFilter*nrow*ncol;
    float* imageR;
    imageR = new float[nElem];

    for(int c=0; c<CHN; c++){
        for (int n=0; n<nFilter; n++){
            for (int i=0; i<nrow; i++){
                for (int j=0; j<ncol; j++){
                    imageR[idx(CHN,nFilter,nrow,ncol,c,n,i,j)] = 0.0;

                    int id = i*ncol + j;
                    float tmp;
                    if (id<0 ||id>ncol*nrow-1){
                        tmp = 0.0f;
                    }else{
                        id += c*nrow*ncol;
                        tmp = h_img[id];
                    }
                    imageR[idx(CHN,nFilter,nrow,ncol,c,n,i,j)] = tmp;
                }
            }
        }
    }

    float* test;
    nElem = CHN*nFilter*nrow*ncol;
    test = new float[nElem];
    hipMemcpy(test, d_imgR, nElem*sizeof(float), hipMemcpyDeviceToHost);

    for (int c=0; c<CHN; c++){
    for (int n=0; n<nFilter; n++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                int np = idx(CHN, nFilter, nrow, ncol, c, n, i, j);
                if ( abs(test[np] - imageR[np]) > 0.001 ) {
                    cout << n << ' ' << c << ' ' << i << ' ' << j << ' ' << test[np] << ' ' << imageR[np] << endl;
                    exit(0);
                };
            }
        }
    }
    }


/*
    for (int c=0; c<CHN; c++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                imageFloat.at<Vec3f>(i,j)[c] = imageR[idx(CHN,nFilter,nrow,ncol,c,0,i,j)]/20.;
            }
        }
    }
    namedWindow("Display Image", WINDOW_AUTOSIZE );
    imshow("Display Image", imageFloat);
    waitKey(0);
*/

    return 0;
}

