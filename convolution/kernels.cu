#include "hip/hip_runtime.h"
__global__ void padding(int const nchl, int const nrow, int const ncol,int const npad,
        float* img, float* imgPad){

    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;
    int const rowP = nrow + 2*npad;
    int const colP = ncol + 2*npad;

    if ( (i<ncol+2*npad) && (j<nrow+2*npad) ){
    for (int n=0; n<nchl; n++){
        int idxP = idxD(nchl, rowP, colP, n, j, i);
        if ( (i>npad-1 && i<ncol+npad) && (j>npad-1 && j<nrow+npad) ) {
            int idxI = idxD(nchl, nrow, ncol, n, j-npad, i-npad);
            imgPad[idxP] = img[idxI];
        } else {
            imgPad[idxP] = 0.f;
        }
    }
    }
}

__global__ void convl(int const nFilter, int const nchl, int const rowP, int const colP, int const rowF, int const colF, float* imgPad, float* imgR, float* filter){

    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;

    __shared__ float tmpFilter[];
    if (i<colP-colF && j<rowP-colF){
        for (int n=0; n<nFilter; n++){
        for (int c=0; c<nchl; c++){
            for (int jj=threadIdx.y; jj<rowF; jj += blockDim.y){
                for (int ii=threadIdx.x; ii<colF; ii += blockDim.x){
                    int idxF = idxD4(nFilter, nchl, rowF, colF, n, c, jj, ii);
                    tmpFilter[jj*colF+ii] = filter[idxF];
                }
            }
            
            int idxR = idxD4(nFilter, nchl, rowP-rowF, colP-colF, n, c, j, i); 
            imgR[idxR] = 0.0f;
            for (int jj=0; jj<rowF; jj++){
                for (int ii=0; ii<colF; ii++){
                    int idxP = idxD(nchl, rowP, colP, c, j+jj, i+ii);
                    imgR[idxR] += imgPad[idxP]*tmpFilter[jj*colF+ii]; 
                }
            } 
        }
        }
    }
}
