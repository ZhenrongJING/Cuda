
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel(int i){

    printf("hello world %d \n", i);
};

int main(){

    int const n_stream = 5;
    hipStream_t *ls_stream;
    ls_stream = (hipStream_t*) new hipStream_t[n_stream];

    for (int i=0; i<n_stream; i++){
        hipStreamCreate(&ls_stream[i]);
    }


    for(int i=0; i<n_stream; i++){
        kernel<<<1, 1, 0, ls_stream[i]>>>(i);
    }

    hipDeviceSynchronize();

    for(int i=0; i<n_stream; i++){
        hipStreamDestroy(ls_stream[i]);
    }

    return 0;
}
