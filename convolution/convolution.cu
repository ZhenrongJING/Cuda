#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <opencv2/opencv.hpp>

#define ROW_F 17
#define COL_F 17

#include "index.hpp"
#include "kernels.cu"
using namespace cv;
using namespace std;


int main(int argc, char** argv )
{
    if ( argc != 2 )
    {
        printf("usage: DisplayImage.out <Image_Path>\n");
        return -1;
    }

    Mat image;
    image = imread( argv[1], 1 );
    Mat imageFloat;
    image.convertTo(imageFloat, CV_32FC3, 1.0/255.0);

    if ( !image.data )
    {
        printf("No image data \n");
        return -1;
    }

    printf( " image type %d\n", image.type() );
    printf( " image type %d\n", imageFloat.type() );

    int const nrow = imageFloat.rows;
    int const ncol = imageFloat.cols;
    int const nchl = imageFloat.channels();
    printf( " size of the image %d times  %d times  %d\n", nchl, nrow, ncol);

    float *h_img;
    h_img = new float[nchl*nrow*ncol];

    int np = 0;
    for (int c=0; c<nchl; c++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                np = c*(nrow*ncol)+i*ncol+j;
                h_img[np] = imageFloat.at<Vec3f>(i,j)[c]; 
            }
        }
    }

    int nElem;
    int const nFilter=2;
    nElem = nFilter*nchl*ROW_F*COL_F;
    float *h_filter;
    h_filter = new float[nElem];

    for (int n=0; n<nFilter; n++){
        for(int c=0; c<nchl; c++){
            for (int i=0; i<ROW_F; i++){
                for (int j=0; j<COL_F; j++){
                    h_filter[idx(nFilter,nchl,ROW_F,COL_F,n,c,i,j)] = rand()/(RAND_MAX+0.f); 
                }
            } 
        }
    }

    float* d_img;
    nElem = nchl*nrow*ncol;
    hipMalloc((void**)&d_img, nElem*sizeof(float));

    float* d_filter;
    nElem = nFilter*nchl*ROW_F*COL_F;
    hipMalloc((void**)&d_filter, nElem*sizeof(float));

    float* d_imgR;
    nElem = nFilter*nchl*nrow*ncol;
    hipMalloc((void**)&d_imgR, nElem*sizeof(float));

    for (int n=0;n<nchl;n++){
        int stride = n*nrow*ncol;
        hipMemcpy(h_img+stride, d_img+stride, (nrow*ncol)*sizeof(float), hipMemcpyHostToDevice);
    }

    float* test;
    nElem = nchl*nrow*ncol;
    test = new float[nElem];
    hipMemcpy(test, d_img, nElem*sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<nElem; i++){
        if ( abs(test[i] - h_img[i]) > 0.0001f ) {
            cout << i << ' ' << test[i] << h_img[i] << endl;
            exit(0);
        }
    }


/*

    gridX = colR/32 + 1;
    gridY = rowR/32 + 1;
    dim3 grid1(gridX,gridY);

    nElem = nFilter*nchl*rowR*colR;
    float* imageR;
    imageR = new float[nElem];

    for (int n=0; n<nFilter; n++){
        for(int c=0; c<nchl; c++){
            for (int i=0; i<rowR; i++){
                for (int j=0; j<colR; j++){

                    imageR[idx(nFilter,nchl,rowR,colR,n,c,i,j)] = 0.0;
                    for(int ii=0; ii<rowF; ii++){
                        for(int jj=0; jj<colF; jj++){
                            imageR[idx(nFilter,nchl,rowR,colR,n,c,i,j)] +=
                                h_filter[idx(nFilter,nchl,rowF,colF,n,c,ii,jj)]
                                *h_imgPad[idx(nchl, rowP, colP, c, i+ii, j+jj)];
                        }
                    }

                }
            }
        }
    }


    for (int n=0; n<nFilter; n++){
    for (int c=0; c<nchl; c++){
        for (int i=0; i<rowR; i++){
            for (int j=0; j<colR; j++){
                int np = idx(nFilter, nchl, rowR, colR, n, c, i, j);
                if ( abs(test[np] - imageR[np]) > 0.001 ) {
                    cout << n << ' ' << c << ' ' << i << ' ' << j << ' ' << test[np] << ' ' << imageR[np] << endl;
                    exit(0);
                };
            }
        }
    }
    }

    for (int c=0; c<nchl; c++){
        for (int i=0; i<nrow; i++){
            for (int j=0; j<ncol; j++){
                imageFloat.at<Vec3f>(i,j)[c] = imageR[idx(nFilter,nchl,rowR,colR,0,c,i,j)]/250.;
            }
        }
    }
    namedWindow("Display Image", WINDOW_AUTOSIZE );
    imshow("Display Image", imageFloat);
    waitKey(0);

*/
    return 0;
}

